#include <cstdio>
#include "vector_arithmetic.cuh"
#include <vector>

hipGraph_t buildQuadraticExpGraph(float *a, float *b, float *c, float *sol1, float *sol2, float *tmp, int numElements, int blockSize) {
    int gridSize = numElements / blockSize + 1;

    hipGraph_t quadraticGraph;
    hipGraphCreate(&quadraticGraph, 0);

    // Build the nodes that are part of the graph
    hipGraphNode_t bSquaredNode;
    hipKernelNodeParams bSquaredParams = {0};
    bSquaredParams.blockDim = blockSize;
    bSquaredParams.gridDim = gridSize;
    bSquaredParams.func = (void *)elementwiseProduct;
    void *bSquaredfunc_params[4] = {(void *)&b, (void *)&b, (void *)&sol1, (void *) &numElements};
    bSquaredParams.kernelParams = (void **)bSquaredfunc_params;

    hipGraphNode_t neg4acNode;
    hipKernelNodeParams neg4acParams = {0};
    neg4acParams.blockDim = blockSize;
    neg4acParams.gridDim = gridSize;
    neg4acParams.func = (void *)elementScalarProduct;
    const float NEG_FOUR = -4.0;
    void *neg4acfunc_params[5] = {(void *)&a, (void *)&c, (void *)&NEG_FOUR, (void *)&sol2, (void *) &numElements};
    neg4acParams.kernelParams = (void **)neg4acfunc_params;

    hipGraphNode_t determinantSumNode;
    hipKernelNodeParams determinantSumParams = {0};
    determinantSumParams.blockDim = blockSize;
    determinantSumParams.gridDim = gridSize;
    determinantSumParams.func = (void *)elementwiseSum;
    void *determinantSumfunc_params[4] = {(void *)&sol1, (void *)&sol2, (void *)&sol1, (void *) &numElements};
    determinantSumParams.kernelParams = (void **)determinantSumfunc_params;

    hipGraphNode_t determinantSqrtNode;
    hipKernelNodeParams determinantSqrtParams = {0};
    determinantSqrtParams.blockDim = blockSize;
    determinantSqrtParams.gridDim = gridSize;
    determinantSqrtParams.func = (void *)elementwiseSqrt;
    void *determinantSqrtfunc_params[3] = {(void *)&sol1, (void *)&tmp, (void *) &numElements};
    determinantSqrtParams.kernelParams = (void **)determinantSqrtfunc_params;
    
    hipGraphNode_t bPlusNode;
    hipKernelNodeParams bPlusParams = {0};
    bPlusParams.blockDim = blockSize;
    bPlusParams.gridDim = gridSize;
    bPlusParams.func = (void *)elementwiseSum;
    void *bPlusfunc_params[4] = {(void *)&tmp, (void *)&b, (void *)&sol1, (void *) &numElements};
    bPlusParams.kernelParams = (void **)bPlusfunc_params;
    
    hipGraphNode_t bMinusNode;
    hipKernelNodeParams bMinusParams = {0};
    bMinusParams.blockDim = blockSize;
    bMinusParams.gridDim = gridSize;
    bMinusParams.func = (void *)elementwiseDifference;
    void *bMinusfunc_params[4] = {(void *)&b, (void *)&tmp,  (void *)&sol2, (void *) &numElements};
    bMinusParams.kernelParams = (void **)bMinusfunc_params;

    hipGraphNode_t bPlusQuotientNode;
    hipKernelNodeParams bPlusQuotientParams = {0};
    bPlusQuotientParams.blockDim = blockSize;
    bPlusQuotientParams.gridDim = gridSize;
    bPlusQuotientParams.func = (void *)elementwiseQuotient;
    const float ONE_HALF = 0.5;
    void *bPlusQuotientfunc_params[5] = {(void *)&sol1, (void *)&a, (void *)&ONE_HALF, (void *)&sol1, (void *) &numElements};
    bPlusQuotientParams.kernelParams = (void **)bPlusQuotientfunc_params;

    hipGraphNode_t bMinusQuotientNode;
    hipKernelNodeParams bMinusQuotientParams = {0};
    bMinusQuotientParams.blockDim = blockSize;
    bMinusQuotientParams.gridDim = gridSize;
    bMinusQuotientParams.func = (void *)elementwiseQuotient;
    void *bMinusQuotientfunc_params[5] = {(void *)&sol2, (void *)&a, (void *)&ONE_HALF, (void *)&sol2, (void *) &numElements};
    bMinusQuotientParams.kernelParams = (void **)bMinusQuotientfunc_params;

    std::vector<hipGraphNode_t> nodeDependencies;

    hipGraphAddKernelNode(&bSquaredNode, quadraticGraph, NULL, 0, &bSquaredParams);
    nodeDependencies.push_back(bSquaredNode);

    hipGraphAddKernelNode(&neg4acNode, quadraticGraph, NULL, 0, &neg4acParams);
    nodeDependencies.push_back(neg4acNode);

    hipGraphAddKernelNode(&determinantSumNode, quadraticGraph, nodeDependencies.data(), 2, &determinantSumParams);

    nodeDependencies.clear();
    nodeDependencies.push_back(determinantSumNode);

    hipGraphAddKernelNode(&determinantSqrtNode, quadraticGraph, nodeDependencies.data(), 1, &determinantSqrtParams);

    nodeDependencies.clear();
    nodeDependencies.push_back(determinantSqrtNode);


    hipGraphAddKernelNode(&bPlusNode, quadraticGraph, nodeDependencies.data(), 1, &bPlusParams);


    hipGraphAddKernelNode(&bMinusNode, quadraticGraph, nodeDependencies.data(), 1, &bMinusParams);

    nodeDependencies.clear();
    nodeDependencies.push_back(bPlusNode);

    hipGraphAddKernelNode(&bPlusQuotientNode, quadraticGraph, nodeDependencies.data(), 1, &bPlusQuotientParams);

    nodeDependencies.clear();
    nodeDependencies.push_back(bMinusNode);

    hipGraphAddKernelNode(&bMinusQuotientNode, quadraticGraph, nodeDependencies.data(), 1, &bMinusQuotientParams);

    return quadraticGraph;
}

hipGraph_t buildQuadraticCapGraph(float *a, float *b, float *c, float *sol1, float *sol2, float *tmp, int numElements, int blockSize) {
    int gridSize = numElements / blockSize + 1;

    hipStream_t bMinus;
    hipStream_t bPlus;
    hipStreamCreate(&bMinus);
    hipStreamCreate(&bPlus);

    hipEvent_t bPlusComplete;
    hipEvent_t bMinusComplete;
    hipEventCreate(&bPlusComplete);
    hipEventCreate(&bMinusComplete);

    hipGraph_t quadraticGraph;
    hipGraphCreate(&quadraticGraph, 0);

    hipStreamBeginCapture(bMinus, hipStreamCaptureModeGlobal);
        // Fork into bPlus to make stream capture record bPlus activity
        hipEventRecord(bMinusComplete, bMinus);
        hipStreamWaitEvent(bPlus, bMinusComplete);

        // Start graph activities
        elementwiseProduct<<<gridSize, blockSize, 0, bMinus>>>(b, b, sol1, numElements);
        elementScalarProduct<<<gridSize, blockSize, 0, bPlus>>>(a, c, -4, sol2, numElements);
        hipEventRecord(bMinusComplete, bMinus);

        hipStreamWaitEvent(bPlus, bMinusComplete);
        elementwiseSum<<<gridSize, blockSize, 0, bPlus>>>(sol1, sol2, sol1, numElements);

        elementwiseSqrt<<<gridSize, blockSize, 0, bPlus>>>(sol1, tmp, numElements);
        hipEventRecord(bPlusComplete, bPlus);

        hipStreamWaitEvent(bMinus, bPlusComplete);
        elementwiseDifference<<<gridSize, blockSize, 0, bMinus>>>(b, tmp, sol1, numElements);
        elementwiseSum<<<gridSize, blockSize, 0, bPlus>>>(b, tmp, sol2, numElements);

        elementwiseQuotient<<<gridSize, blockSize, 0, bMinus>>>(sol1, a, 0.5, sol1, numElements);
        elementwiseQuotient<<<gridSize, blockSize, 0, bPlus>>>(sol2, a, 0.5, sol2, numElements);

        // Join the bPlus stream back into bMinus
        hipEventRecord(bPlusComplete, bPlus);
        hipStreamWaitEvent(bMinus, bPlusComplete);

    hipStreamEndCapture(bMinus, &quadraticGraph);

    hipEventDestroy(bPlusComplete);
    hipEventDestroy(bMinusComplete);

    hipStreamDestroy(bMinus);
    hipStreamDestroy(bPlus);

    return quadraticGraph;
}

float timeGraph(hipGraph_t graph, int iterations) {
    hipEvent_t clockStart, clockStop;
    hipEventCreate(&clockStart);
    hipEventCreate(&clockStop);

    hipGraphExec_t executable;
    
    hipGraphInstantiate(&executable, graph, NULL, NULL, 0);

    hipEventRecord(clockStart);
        for(int i=0; i<iterations; i++)
        hipGraphLaunch(executable, 0);
    hipEventRecord(clockStop);
    hipEventSynchronize(clockStop);

    float timeElapsed;
    hipEventElapsedTime(&timeElapsed, clockStart, clockStop);

    hipEventDestroy(clockStart);
    hipEventDestroy(clockStop);

    return timeElapsed;
}

float quadraticUsingEvents(float *a, float *b, float *c, float *sol1, float *sol2, float* tmp, int numElements, int blockSize, int iterations) {
    int gridSize = (numElements / blockSize) + 1;

    hipStream_t bMinus;
    hipStream_t bPlus;
    hipStreamCreate(&bMinus);
    hipStreamCreate(&bPlus);

    hipEvent_t bPlusComplete, bMinusComplete;
    hipEventCreate(&bPlusComplete);
    hipEventCreate(&bMinusComplete);

    hipEvent_t clockStart, clockStop;
    hipEventCreate(&clockStart);
    hipEventCreate(&clockStop);

    // Warm up both streams before beginning timing
    elementwiseProduct<<<gridSize, blockSize, 0, bMinus>>>(b, b, sol1, numElements);
    elementScalarProduct<<<gridSize, blockSize, 0, bPlus>>>(a, c, -4, sol2, numElements);
    hipStreamSynchronize(bMinus);
    hipStreamSynchronize(bPlus);

    hipEventRecord(clockStart);

        for (int i=0; i<iterations; i++) { 

            // Concurrent
            elementwiseProduct<<<gridSize, blockSize, 0, bMinus>>>(b, b, sol1, numElements);
            elementScalarProduct<<<gridSize, blockSize, 0, bPlus>>>(a, c, -4, sol2, numElements);

            // Use events to ensure completion
            hipEventRecord(bMinusComplete, bMinus);
            hipStreamWaitEvent(bPlus, bMinusComplete);

            elementwiseSum<<<gridSize, blockSize, 0, bPlus>>>(sol1, sol2, sol1, numElements);
            elementwiseSqrt<<<gridSize, blockSize, 0, bPlus>>>(sol1, tmp, numElements);

            // Sync again - must have determinant before proceeding
            hipEventRecord(bPlusComplete, bPlus);
            hipStreamWaitEvent(bMinus, bPlusComplete);

            elementwiseDifference<<<gridSize, blockSize, 0, bMinus>>>(b, tmp, sol1, numElements);
            elementwiseSum<<<gridSize, blockSize, 0, bPlus>>>(b, tmp, sol2, numElements);

            elementwiseQuotient<<<gridSize, blockSize, 0, bMinus>>>(sol1, a, 0.5, sol1, numElements);
            elementwiseQuotient<<<gridSize, blockSize, 0, bPlus>>>(sol2, a, 0.5, sol2, numElements);
        }

    // Make sure that both streams are done before stopping timer
    hipEventRecord(bMinusComplete, bMinus);
    hipEventRecord(bPlusComplete, bPlus);
    hipEventSynchronize(bMinusComplete);
    hipEventSynchronize(bPlusComplete);

    hipEventRecord(clockStop);
    hipEventSynchronize(clockStop);
    float timeElapsed;
    hipEventElapsedTime(&timeElapsed, clockStart, clockStop);

    hipEventDestroy(clockStart);
    hipEventDestroy(clockStop);

    hipStreamDestroy(bMinus);
    hipStreamDestroy(bPlus);

    return timeElapsed;
}

float synchronousQuadratic(float *a, float *b, float *c, float *sol1, float *sol2, float *tmp, int numElements, int blockSize, int iterations) {
    int gridSize = (numElements / blockSize) + 1;

    hipEvent_t clockStart, clockStop;
    hipEventCreate(&clockStart);
    hipEventCreate(&clockStop);

    // Warm up before beginning timing
    elementwiseProduct<<<gridSize, blockSize>>>(b, b, sol1, numElements);

    hipEventRecord(clockStart);

        for (int i=0; i<iterations; i++) { 

            elementwiseProduct<<<gridSize, blockSize>>>(b, b, sol1, numElements);
            elementScalarProduct<<<gridSize, blockSize>>>(a, c, -4, sol2, numElements);

            elementwiseSum<<<gridSize, blockSize>>>(sol1, sol2, sol1, numElements);
            elementwiseSqrt<<<gridSize, blockSize>>>(sol1, tmp, numElements);

            elementwiseDifference<<<gridSize, blockSize>>>(b, tmp, sol1, numElements);
            elementwiseSum<<<gridSize, blockSize>>>(b, tmp, sol2, numElements);

            elementwiseQuotient<<<gridSize, blockSize>>>(sol1, a, 0.5, sol1, numElements);
            elementwiseQuotient<<<gridSize, blockSize>>>(sol2, a, 0.5, sol2, numElements);
        }

    hipEventRecord(clockStop);
    hipEventSynchronize(clockStop);
    float timeElapsed;
    hipEventElapsedTime(&timeElapsed, clockStart, clockStop);

    hipEventDestroy(clockStart);
    hipEventDestroy(clockStop);

    return timeElapsed;
}

int main(int argc, char *argv[]) {
    const int NUM_ELEMENTS = 64;
    const int BLOCK_SIZE = 32;
    const int NUM_TRIALS = 20;
    const int ITERATIONS = 131072;

    int gridSize = NUM_ELEMENTS / BLOCK_SIZE + 1;

    float *a, *b, *c, *sol1, *sol2, *tmp;
    hipMalloc(&a, sizeof(float)*NUM_ELEMENTS);
    hipMalloc(&b, sizeof(float)*NUM_ELEMENTS);
    hipMalloc(&c, sizeof(float)*NUM_ELEMENTS);
    hipMalloc(&sol1, sizeof(float)*NUM_ELEMENTS);
    hipMalloc(&sol2, sizeof(float)*NUM_ELEMENTS);
    hipMalloc(&tmp, sizeof(float)*NUM_ELEMENTS);

    fillArray<<<gridSize, BLOCK_SIZE>>>(a, NUM_ELEMENTS);
    fillArray<<<gridSize, BLOCK_SIZE>>>(b, NUM_ELEMENTS);
    fillArray<<<gridSize, BLOCK_SIZE>>>(c, NUM_ELEMENTS);

    hipGraph_t expGraph = buildQuadraticExpGraph(a, b, c,sol1, sol2, tmp, NUM_ELEMENTS, BLOCK_SIZE);
    hipGraph_t capGraph = buildQuadraticCapGraph(a, b, c,sol1, sol2, tmp, NUM_ELEMENTS, BLOCK_SIZE); 

    printf("Synchronous,Stream,Captured Graph,Explicit Graph\n");
    for (int i=0; i<NUM_TRIALS; i++) {
        float syncTime = synchronousQuadratic(a, b, c, sol1, sol2, tmp, NUM_ELEMENTS, BLOCK_SIZE, ITERATIONS);
        printf("%.4f,", syncTime);
        float streamTime = quadraticUsingEvents(a, b, c, sol1, sol2, tmp, NUM_ELEMENTS, BLOCK_SIZE, ITERATIONS);
        printf("%.4f,", streamTime);
        float capGraphTime = timeGraph(capGraph, ITERATIONS);
        printf("%.4f,", capGraphTime);
        float expGraphTime = timeGraph(expGraph, ITERATIONS);
        printf("%.4f\n", expGraphTime);
    }

    hipGraphDestroy(expGraph);
    hipGraphDestroy(capGraph);

    hipFree(a);
    hipFree(b);
    hipFree(c);
    hipFree(tmp);
    hipFree(sol1);
    hipFree(sol2);
}