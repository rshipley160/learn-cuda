
#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void array_fill_1D(int *array, int arraySize, int value) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    if (id < arraySize)
        array[id] = value;
}

int main(int argc, char *argv[]) {
    const int NUM_ELEMENTS = 32;
    int block_size = 32;
    int grid_size = 1;

    int *h_array;
    hipHostMalloc(&h_array, sizeof(int)*NUM_ELEMENTS, hipHostMallocDefault);

    int *d_array;
    hipMalloc(&d_array, sizeof(int)*NUM_ELEMENTS);

    array_fill_1D<<<grid_size,block_size>>>(d_array, NUM_ELEMENTS, 1);

    hipStream_t stream;
    hipStreamCreate(&stream);

    hipMemcpyAsync(h_array, d_array, sizeof(int)*NUM_ELEMENTS, hipMemcpyDeviceToHost, stream);

    hipStreamSynchronize(stream);

    for(int i=0; i<NUM_ELEMENTS; i++)
        printf("%d ",h_array[i]);

    hipStreamDestroy(stream);
    hipFree(d_array);
    hipHostFree(h_array);
}
