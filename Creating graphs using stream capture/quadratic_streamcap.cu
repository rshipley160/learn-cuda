#include <cstdio>
#include "vector_arithmetic.cuh"

int main(int argc, char *argv[]) {
    const int NUM_ELEMENTS = 64;
    const int BLOCK_SIZE = 32;
    int gridSize = NUM_ELEMENTS / BLOCK_SIZE + 1;

    float *a, *b, *c, *sol1, *sol2, *tmp;
    hipMalloc(&a, sizeof(float)*NUM_ELEMENTS);
    hipMalloc(&b, sizeof(float)*NUM_ELEMENTS);
    hipMalloc(&c, sizeof(float)*NUM_ELEMENTS);
    hipMalloc(&sol1, sizeof(float)*NUM_ELEMENTS);
    hipMalloc(&sol2, sizeof(float)*NUM_ELEMENTS);
    hipMalloc(&tmp, sizeof(float)*NUM_ELEMENTS);

    fillArray<<<gridSize, BLOCK_SIZE>>>(a, NUM_ELEMENTS, 1);
    fillArray<<<gridSize, BLOCK_SIZE>>>(b, NUM_ELEMENTS, 2);
    fillArray<<<gridSize, BLOCK_SIZE>>>(c, NUM_ELEMENTS, 1);

    hipStream_t bMinus;
    hipStream_t bPlus;
    hipStreamCreate(&bMinus);
    hipStreamCreate(&bPlus);

    hipEvent_t bPlusComplete;
    hipEvent_t bMinusComplete;
    hipEventCreate(&bPlusComplete);
    hipEventCreate(&bMinusComplete);

    hipGraph_t quadraticGraph;
    hipGraphCreate(&quadraticGraph, 0);

    hipStreamBeginCapture(bMinus, hipStreamCaptureModeGlobal);
        // Fork into bPlus to make stream capture record bPlus activity
        hipEventRecord(bMinusComplete, bMinus);
        hipStreamWaitEvent(bPlus, bMinusComplete);

        // Start graph activities
        elementwiseProduct<<<gridSize, BLOCK_SIZE, 0, bMinus>>>(b, b, sol1, NUM_ELEMENTS);
        elementScalarProduct<<<gridSize, BLOCK_SIZE, 0, bPlus>>>(a, c, -4, sol2, NUM_ELEMENTS);
        hipEventRecord(bMinusComplete, bMinus);

        hipStreamWaitEvent(bPlus, bMinusComplete);
        elementwiseSum<<<gridSize, BLOCK_SIZE, 0, bPlus>>>(sol1, sol2, sol1, NUM_ELEMENTS);

        elementwiseSqrt<<<gridSize, BLOCK_SIZE, 0, bPlus>>>(sol1, tmp, NUM_ELEMENTS);
        hipEventRecord(bPlusComplete, bPlus);

        hipStreamWaitEvent(bMinus, bPlusComplete);
        elementwiseDifference<<<gridSize, BLOCK_SIZE, 0, bMinus>>>(b, tmp, sol1, NUM_ELEMENTS);
        elementwiseSum<<<gridSize, BLOCK_SIZE, 0, bPlus>>>(b, tmp, sol2, NUM_ELEMENTS);

        elementwiseQuotient<<<gridSize, BLOCK_SIZE, 0, bMinus>>>(sol1, a, 0.5, sol1, NUM_ELEMENTS);
        elementwiseQuotient<<<gridSize, BLOCK_SIZE, 0, bPlus>>>(sol2, a, 0.5, sol2, NUM_ELEMENTS);
        hipEventRecord(bPlusComplete, bPlus);
        hipStreamWaitEvent(bMinus, bPlusComplete);

    hipStreamEndCapture(bMinus, &quadraticGraph);

    hipEventDestroy(bPlusComplete);
    hipEventDestroy(bMinusComplete);

    hipGraphExec_t graphExecutable;
    hipGraphInstantiate(&graphExecutable, quadraticGraph, NULL, NULL, 0);

    hipStreamDestroy(bMinus);
    hipStreamDestroy(bPlus);

    hipStream_t newStream;
    hipStreamCreate(&newStream);
    
    hipGraphLaunch(graphExecutable, newStream);
    hipStreamSynchronize(newStream);

    hipStreamDestroy(newStream);

    hipFree(a);
    hipFree(b);
    hipFree(c);
    hipFree(tmp);
    hipFree(sol1);
    hipFree(sol2);
}