#include <cstdio>
#include "vector_arithmetic.cuh"

#define BLOCK_SIZE 512

float quadraticUsingEvents(int numElements, int iterations) {
    int gridSize = (numElements / BLOCK_SIZE) + 1;

    float *a, *b, *c, *sol1, *sol2, *tmp;
    hipMalloc(&a, sizeof(float)*numElements);
    hipMalloc(&b, sizeof(float)*numElements);
    hipMalloc(&c, sizeof(float)*numElements);
    hipMalloc(&sol1, sizeof(float)*numElements);
    hipMalloc(&sol2, sizeof(float)*numElements);
    hipMalloc(&tmp, sizeof(float)*numElements);

    fillArray<<<gridSize, BLOCK_SIZE>>>(a, numElements);
    fillArray<<<gridSize, BLOCK_SIZE>>>(b, numElements);
    fillArray<<<gridSize, BLOCK_SIZE>>>(c, numElements);

    hipStream_t bMinus;
    hipStream_t bPlus;
    hipStreamCreate(&bMinus);
    hipStreamCreate(&bPlus);

    hipEvent_t bPlusComplete, bMinusComplete;
    hipEventCreate(&bPlusComplete);
    hipEventCreate(&bMinusComplete);

    hipEvent_t clockStart, clockStop;
    hipEventCreate(&clockStart);
    hipEventCreate(&clockStop);

    // Warm up both streams before beginning timing
    elementwiseProduct<<<gridSize, BLOCK_SIZE, 0, bMinus>>>(b, b, sol1, numElements);
    elementScalarProduct<<<gridSize, BLOCK_SIZE, 0, bPlus>>>(a, c, -4, sol2, numElements);
    hipStreamSynchronize(bMinus);
    hipStreamSynchronize(bPlus);

    hipEventRecord(clockStart);

        for (int i=0; i<iterations; i++) { 

        // Concurrent
        elementwiseProduct<<<gridSize, BLOCK_SIZE, 0, bMinus>>>(b, b, sol1, numElements);
        elementScalarProduct<<<gridSize, BLOCK_SIZE, 0, bPlus>>>(a, c, -4, sol2, numElements);

        // Use events to ensure completion
        hipEventRecord(bMinusComplete, bMinus);
        hipStreamWaitEvent(bPlus, bMinusComplete);

        elementwiseSum<<<gridSize, BLOCK_SIZE, 0, bPlus>>>(sol1, sol2, sol1, numElements);
        elementwiseSqrt<<<gridSize, BLOCK_SIZE, 0, bPlus>>>(sol1, tmp, numElements);

        // Sync again - must have determinant before proceeding
        hipEventRecord(bPlusComplete, bPlus);
        hipStreamWaitEvent(bMinus, bPlusComplete);

        elementwiseDifference<<<gridSize, BLOCK_SIZE, 0, bMinus>>>(b, tmp, sol1, numElements);
        elementwiseSum<<<gridSize, BLOCK_SIZE, 0, bPlus>>>(b, tmp, sol2, numElements);

        elementwiseQuotient<<<gridSize, BLOCK_SIZE, 0, bMinus>>>(sol1, a, 0.5, sol1, numElements);
        elementwiseQuotient<<<gridSize, BLOCK_SIZE, 0, bPlus>>>(sol2, a, 0.5, sol2, numElements);

        // Make sure that both streams are done before stopping timer
        hipEventRecord(bMinusComplete, bMinus);
        hipEventRecord(bPlusComplete, bPlus);
        hipEventSynchronize(bMinusComplete);
        hipEventSynchronize(bPlusComplete);

        }

    hipEventRecord(clockStop);
    hipEventSynchronize(clockStop);
    float timeElapsed;
    hipEventElapsedTime(&timeElapsed, clockStart, clockStop);

    hipEventDestroy(clockStart);
    hipEventDestroy(clockStop);

    hipStreamDestroy(bMinus);
    hipStreamDestroy(bPlus);

    hipFree(a);
    hipFree(b);
    hipFree(c);
    hipFree(tmp);
    hipFree(sol1);
    hipFree(sol2);

    return timeElapsed;
}

float quadraticUsingStreamSync(int numElements, int iterations) {
    int gridSize = numElements / BLOCK_SIZE + 1;

    float *a, *b, *c, *sol1, *sol2, *tmp;
    hipMalloc(&a, sizeof(float)*numElements);
    hipMalloc(&b, sizeof(float)*numElements);
    hipMalloc(&c, sizeof(float)*numElements);
    hipMalloc(&sol1, sizeof(float)*numElements);
    hipMalloc(&sol2, sizeof(float)*numElements);
    hipMalloc(&tmp, sizeof(float)*numElements);

    fillArray<<<gridSize, BLOCK_SIZE>>>(a, numElements);
    fillArray<<<gridSize, BLOCK_SIZE>>>(b, numElements);
    fillArray<<<gridSize, BLOCK_SIZE>>>(c, numElements);

    hipStream_t bMinus;
    hipStream_t bPlus;
    hipStreamCreate(&bMinus);
    hipStreamCreate(&bPlus);

    hipEvent_t clockStart, clockStop;
    hipEventCreate(&clockStart);
    hipEventCreate(&clockStop);

    // Warm up both streams before beginning timing
    elementwiseProduct<<<gridSize, BLOCK_SIZE, 0, bMinus>>>(b, b, sol1, numElements);
    elementScalarProduct<<<gridSize, BLOCK_SIZE, 0, bPlus>>>(a, c, -4, sol2, numElements);
    hipStreamSynchronize(bMinus);
    hipStreamSynchronize(bPlus);

    hipEventRecord(clockStart);

        for (int i=0; i<iterations; i++) { 

            // Concurrent
            elementwiseProduct<<<gridSize, BLOCK_SIZE, 0, bMinus>>>(b, b, sol1, numElements);
            elementScalarProduct<<<gridSize, BLOCK_SIZE, 0, bPlus>>>(a, c, -4, sol2, numElements);

            // Sync streams to ensure these complete before next step
            hipStreamSynchronize(bMinus);
            hipStreamSynchronize(bPlus);

            elementwiseSum<<<gridSize, BLOCK_SIZE, 0, bPlus>>>(sol1, sol2, sol1, numElements);

            elementwiseSqrt<<<gridSize, BLOCK_SIZE, 0, bPlus>>>(sol1, tmp, numElements);

            // Sync again - must have determinant before proceeding
            hipStreamSynchronize(bPlus);

            elementwiseDifference<<<gridSize, BLOCK_SIZE, 0, bMinus>>>(b, tmp, sol1, numElements);
            elementwiseSum<<<gridSize, BLOCK_SIZE, 0, bPlus>>>(b, tmp, sol2, numElements);

            elementwiseQuotient<<<gridSize, BLOCK_SIZE, 0, bMinus>>>(sol1, a, 0.5, sol1, numElements);
            elementwiseQuotient<<<gridSize, BLOCK_SIZE, 0, bPlus>>>(sol2, a, 0.5, sol2, numElements);

            // Make sure that both streams are done before stopping timer
            hipStreamSynchronize(bPlus);
            hipStreamSynchronize(bMinus);

        }

    hipEventRecord(clockStop);
    hipEventSynchronize(clockStop);
    float timeElapsed;
    hipEventElapsedTime(&timeElapsed, clockStart, clockStop);

    hipEventDestroy(clockStart);
    hipEventDestroy(clockStop);

    hipStreamDestroy(bMinus);
    hipStreamDestroy(bPlus);

    hipFree(a);
    hipFree(b);
    hipFree(c);
    hipFree(tmp);
    hipFree(sol1);
    hipFree(sol2);

    return timeElapsed;
}

float quadraticUsingDeviceSync(int numElements, int iterations) {
    int gridSize = numElements / BLOCK_SIZE + 1;

    float *a, *b, *c, *sol1, *sol2, *tmp;
    hipMalloc(&a, sizeof(float)*numElements);
    hipMalloc(&b, sizeof(float)*numElements);
    hipMalloc(&c, sizeof(float)*numElements);
    hipMalloc(&sol1, sizeof(float)*numElements);
    hipMalloc(&sol2, sizeof(float)*numElements);
    hipMalloc(&tmp, sizeof(float)*numElements);

    fillArray<<<gridSize, BLOCK_SIZE>>>(a, numElements);
    fillArray<<<gridSize, BLOCK_SIZE>>>(b, numElements);
    fillArray<<<gridSize, BLOCK_SIZE>>>(c, numElements);

    hipStream_t bMinus;
    hipStream_t bPlus;
    hipStreamCreate(&bMinus);
    hipStreamCreate(&bPlus);

    hipEvent_t clockStart, clockStop;
    hipEventCreate(&clockStart);
    hipEventCreate(&clockStop);

    // Warm up both streams before beginning timing
    elementwiseProduct<<<gridSize, BLOCK_SIZE, 0, bMinus>>>(b, b, sol1, numElements);
    elementScalarProduct<<<gridSize, BLOCK_SIZE, 0, bPlus>>>(a, c, -4, sol2, numElements);
    hipStreamSynchronize(bMinus);
    hipStreamSynchronize(bPlus);

    hipEventRecord(clockStart);

        for (int i=0; i<iterations; i++) { 
            // Concurrent
            elementwiseProduct<<<gridSize, BLOCK_SIZE, 0, bMinus>>>(b, b, sol1, numElements);
            elementScalarProduct<<<gridSize, BLOCK_SIZE, 0, bPlus>>>(a, c, -4, sol2, numElements);

            // Sync device to ensure these complete before next step
            hipDeviceSynchronize();

            elementwiseSum<<<gridSize, BLOCK_SIZE, 0, bPlus>>>(sol1, sol2, sol1, numElements);

            elementwiseSqrt<<<gridSize, BLOCK_SIZE, 0, bPlus>>>(sol1, tmp, numElements);

            // Sync again - must have determinant before proceeding
            hipDeviceSynchronize();

            elementwiseDifference<<<gridSize, BLOCK_SIZE, 0, bMinus>>>(b, tmp, sol1, numElements);
            elementwiseSum<<<gridSize, BLOCK_SIZE, 0, bPlus>>>(b, tmp, sol2, numElements);

            elementwiseQuotient<<<gridSize, BLOCK_SIZE, 0, bMinus>>>(sol1, a, 0.5, sol1, numElements);
            elementwiseQuotient<<<gridSize, BLOCK_SIZE, 0, bPlus>>>(sol2, a, 0.5, sol2, numElements);

            // Make sure that both streams are done before stopping timer
            hipDeviceSynchronize();
        }

    hipEventRecord(clockStop);
    hipEventSynchronize(clockStop);
    float timeElapsed;
    hipEventElapsedTime(&timeElapsed, clockStart, clockStop);

    hipEventDestroy(clockStart);
    hipEventDestroy(clockStop);

    hipStreamDestroy(bMinus);
    hipStreamDestroy(bPlus);

    hipFree(a);
    hipFree(b);
    hipFree(c);
    hipFree(tmp);
    hipFree(sol1);
    hipFree(sol2);

    return timeElapsed;
}

int main(int argc, char *argv[]) {
    const int NUM_ELEMENTS = 16382;
    const int TRIALS = 20;

    printf("Device,Stream,Event\n");
    for (int i=0; i<TRIALS; i++) {
        printf("%.4f,",quadraticUsingDeviceSync(NUM_ELEMENTS, 1024));
        printf("%.4f,",quadraticUsingStreamSync(NUM_ELEMENTS, 1024));
        printf("%.4f\n",quadraticUsingEvents(NUM_ELEMENTS, 1024));
    }
}