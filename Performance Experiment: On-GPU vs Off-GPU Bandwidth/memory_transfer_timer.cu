#include "hip/hip_runtime.h"
%%cu
#include <cstdio>

// GPUs will vary on the max number of threads that can be in a block, but most can accomodate 512 per block
#define THREADS_PER_BLOCK 512

/*
    gpu-bandwidth bytes numTests
    bytes - number of megabytes (MiB) to transfer in each test
    numTests - the number of times to run each bandwidth test
*/

// Generates a unique ID for each thread to ensure array accesses are thread safe
__device__ int globalID()
{
    return blockIdx.x * blockDim.x + threadIdx.x;
}

// Copies contents of one array to another using the GPU
__global__ void knlMemCopy_1D(int *inbound, int *outbound, long size)
{
    int uniqueID = globalID();

    if (uniqueID < size)
        outbound[uniqueID] = inbound[uniqueID];
}

float *kernelCopy(long transfer_size, int repetitions) {
    // Calculate the number of blocks to use - we want one thread per element (byte) copied
    int numBlocks =  transfer_size / THREADS_PER_BLOCK;

    // If there are more bytes than can be evenly divided by a block's worth of threads, add a block to cover the rest
    if (transfer_size % THREADS_PER_BLOCK > 0)
        numBlocks++;

    hipEvent_t clockStart, clockStop;
    hipEventCreate(&clockStart);
    hipEventCreate(&clockStop);

    int *d_input, *d_output;
    hipMalloc(&d_input, transfer_size*sizeof(int));
    hipMalloc(&d_output, transfer_size*sizeof(int));
    

    float trialTime;
    float *results = (float *) malloc(sizeof(float)*repetitions);

    // Initial run through to avoid any cold-start outliers
    knlMemCopy_1D<<<numBlocks, THREADS_PER_BLOCK>>>(d_input, d_output, transfer_size);

    for (int rep=0; rep < repetitions; rep++)
    {
        hipEventRecord(clockStart, 0);

            knlMemCopy_1D<<<numBlocks, THREADS_PER_BLOCK>>>(d_input, d_output, transfer_size);
        
        hipEventRecord(clockStop, 0);
        hipEventSynchronize(clockStop);
        hipEventElapsedTime(&trialTime, clockStart, clockStop);

        results[rep] = trialTime;
    }

    hipFree(d_input);
    hipFree(d_output);

    return results;
}

float *memcpyDtoD(long transfer_size, int repetitions) {
	// Calculate the number of blocks to use - we want one thread per element (byte) copied
    int numBlocks =  transfer_size / THREADS_PER_BLOCK;

    // If there are more bytes than can be evenly divided by a block's worth of threads, add a block to cover the rest
    if (transfer_size % THREADS_PER_BLOCK > 0)
        numBlocks++;

    hipEvent_t clockStart, clockStop;
    hipEventCreate(&clockStart);
    hipEventCreate(&clockStop);

    int *d_input, *d_output;
    hipMalloc(&d_input, transfer_size*sizeof(int));
    hipMalloc(&d_output, transfer_size*sizeof(int));
    

    float trialTime;
    float *results = (float *) malloc(sizeof(float)*repetitions);

    // Initial run through to avoid any cold-start outliers
    hipMemcpy(d_output, d_input, transfer_size*sizeof(int), hipMemcpyDeviceToDevice);

    for (int rep=0; rep < repetitions; rep++)
    {
        hipEventRecord(clockStart, 0);

            hipMemcpy(d_output, d_input, transfer_size*sizeof(int), hipMemcpyDeviceToDevice);
        
        hipEventRecord(clockStop, 0);
        hipEventSynchronize(clockStop);
        hipEventElapsedTime(&trialTime, clockStart, clockStop);

        results[rep] = trialTime;
    }

    hipFree(d_input);
    hipFree(d_output);

    return results;
}

float *pagedDtoH(long transfer_size, int repetitions) {
	// Calculate the number of blocks to use - we want one thread per element (byte) copied
    int numBlocks =  transfer_size / THREADS_PER_BLOCK;

    // If there are more bytes than can be evenly divided by a block's worth of threads, add a block to cover the rest
    if (transfer_size % THREADS_PER_BLOCK > 0)
        numBlocks++;

    hipEvent_t clockStart, clockStop;
    hipEventCreate(&clockStart);
    hipEventCreate(&clockStop);

    int *d_input, *h_output;
    hipMalloc(&d_input, transfer_size*sizeof(int));
    h_output = (int *) malloc(transfer_size*sizeof(int));
    

    float trialTime;
    float *results = (float *) malloc(sizeof(float)*repetitions);

    // Initial run through to avoid any cold-start outliers
    hipMemcpy(h_output, d_input, transfer_size*sizeof(int), hipMemcpyDeviceToDevice);

    for (int rep=0; rep < repetitions; rep++)
    {
        hipEventRecord(clockStart, 0);

            hipMemcpy(h_output, d_input, transfer_size*sizeof(int), hipMemcpyDeviceToHost);
        
        hipEventRecord(clockStop, 0);
        hipEventSynchronize(clockStop);
        hipEventElapsedTime(&trialTime, clockStart, clockStop);

        results[rep] = trialTime;
    }

    hipFree(d_input);
    free(h_output);

    return results;
}

float *pinnedDtoH(long transfer_size, int repetitions) {
	// Calculate the number of blocks to use - we want one thread per element (byte) copied
    int numBlocks =  transfer_size / THREADS_PER_BLOCK;

    // If there are more bytes than can be evenly divided by a block's worth of threads, add a block to cover the rest
    if (transfer_size % THREADS_PER_BLOCK > 0)
        numBlocks++;

    hipEvent_t clockStart, clockStop;
    hipEventCreate(&clockStart);
    hipEventCreate(&clockStop);

    int *d_input, *h_output;
    hipMalloc(&d_input, transfer_size*sizeof(int));
    hipHostMalloc(&h_output, transfer_size*sizeof(int));
    

    float trialTime;
    float *results = (float *) malloc(sizeof(float)*repetitions);

    // Initial run through to avoid any cold-start outliers
    hipMemcpy(h_output, d_input, transfer_size*sizeof(int), hipMemcpyDeviceToDevice);

    for (int rep=0; rep < repetitions; rep++)
    {
        hipEventRecord(clockStart, 0);

            hipMemcpy(h_output, d_input, transfer_size*sizeof(int), hipMemcpyDeviceToHost);
        
        hipEventRecord(clockStop, 0);
        hipEventSynchronize(clockStop);
        hipEventElapsedTime(&trialTime, clockStart, clockStop);

        results[rep] = trialTime;
    }

    hipFree(d_input);
    hipFree(h_output);

    return results;
}

float *unifiedMemcpy(long transfer_size, int repetitions) {
	// Calculate the number of blocks to use - we want one thread per element (byte) copied
    int numBlocks =  transfer_size / THREADS_PER_BLOCK;

    // If there are more bytes than can be evenly divided by a block's worth of threads, add a block to cover the rest
    if (transfer_size % THREADS_PER_BLOCK > 0)
        numBlocks++;

    hipEvent_t clockStart, clockStop;
    hipEventCreate(&clockStart);
    hipEventCreate(&clockStop);

    int *u_input, *u_output;
    hipMallocManaged(&u_input, transfer_size*sizeof(int));
	hipMallocManaged(&u_output, transfer_size*sizeof(int));
    

    float trialTime;
    float *results = (float *) malloc(sizeof(float)*repetitions);

    // Initial run through to avoid any cold-start outliers
	memcpy(u_output, u_input, transfer_size*sizeof(int));

    for (int rep=0; rep < repetitions; rep++)
    {
        hipEventRecord(clockStart, 0);

		memcpy(u_output, u_input, transfer_size*sizeof(int));
        
        hipEventRecord(clockStop, 0);
        hipEventSynchronize(clockStop);
        hipEventElapsedTime(&trialTime, clockStart, clockStop);

        results[rep] = trialTime;
    }

    hipFree(u_input);
    hipFree(u_output);

    return results;
}

int main(int argc, char *argv[])
{
    // if (argc < 3) {
    //     printf("Too few arguments supplied. Make sure to supply the amount of memory to transfer (in MiB) and the number of repetitions to perform.\n");
    //     return 1;
    // } 

    int num_mibibytes = 128;//atoi(argv[1]);
    int repetitions = 10;//atoi(argv[2]);

    const int MI_B = 1048576;    // One MiB or 2^20 bytes

    // Total number of bytes (integer elements) to copy
    long transfer_size = num_mibibytes * MI_B / sizeof(int);

	printf("type,size,unit,numAccesses");
    for (int rep=1; rep <= repetitions; rep++)
        printf(",run%d",rep);
    printf("\n");

    // Test #1: GPU kernel copy capablilty    
    float *kernelCopyTimes = kernelCopy(transfer_size, repetitions);

	printf("kernelCopy,%d,MB,2",num_mibibytes);
    for (int rep=0; rep < repetitions; rep++)
        printf(",%f",kernelCopyTimes[rep]);
    printf("\n");

    free(kernelCopyTimes);

	// Test #2: GPU-GPU memcpy capablilty    
    float *deviceMemcpyTimes = memcpyDtoD(transfer_size, repetitions);

	printf("memcpyDtoD,%d,MB,2",num_mibibytes);
    for (int rep=0; rep < repetitions; rep++)
        printf(",%f",deviceMemcpyTimes[rep]);
    printf("\n");

    free(deviceMemcpyTimes);

	// Test #3: Paged GPU-CPU memcpy capablilty    
    float *pagedMemcpyTimes = pagedDtoH(transfer_size, repetitions);

	printf("pagedDtoH,%d,MB,2",num_mibibytes);
    for (int rep=0; rep < repetitions; rep++)
        printf(",%f",pagedMemcpyTimes[rep]);
    printf("\n");

    free(pagedMemcpyTimes);
 
 	// Test #4: Pinned GPU-CPU memcpy capablilty    
    float *pinnedMemcpyTimes = pinnedDtoH(transfer_size, repetitions);

	printf("pinnedDtoH,%d,MB,2",num_mibibytes);
    for (int rep=0; rep < repetitions; rep++)
        printf(",%f",pinnedMemcpyTimes[rep]);
    printf("\n");

    free(pinnedMemcpyTimes);

	// Test #5: Unified memory copy capablilty    
    float *unifiedMemcpyTimes = unifiedMemcpy(transfer_size, repetitions);

	printf("unifiedMemcpy,%d,MB,2",num_mibibytes);
    for (int rep=0; rep < repetitions; rep++)
        printf(",%f",unifiedMemcpyTimes[rep]);
    printf("\n");

	free(unifiedMemcpyTimes);
}
